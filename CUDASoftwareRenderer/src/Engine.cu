#include<pch.h>
#include"Engine.cuh"
#include"DIB.cuh"
#include"DeviceMemory.cuh"
#include"Color.cuh"
#include"Geometry.cuh"

Engine::Engine(HWND hWnd)
	: mHandle(hWnd), mDIB(std::make_shared<DIB>(hWnd, 1280, 720)),
	mResources(std::make_shared<ResourceManager>()),
	mRenderer(std::make_unique<Renderer>(mDIB, mResources))
{
	mTexture = mResources->CreateTexture2D(1280, 720);

}

void Engine::Start()
{
	//FbxLoader sampleLoader("");

	//unsigned int vertexCount = sampleLoader.Vertices.size();
	//unsigned int indexCount = sampleLoader.Indices.size();
	//
	//std::vector<SampleVertex> vertices;

	//for (unsigned int i = 0; i < vertexCount; i++)
	//{
	//	Vertex conv = sampleLoader.Vertices[i];
	//	vertices.push_back(ConvertVertex(conv));
	//}

	//mVertexBuffer = mResources->CreateBuffer(sizeof(SampleVertex), vertexCount, vertices.data());
	//mIndexBuffer = mResources->CreateBuffer(sizeof(unsigned int), indexCount, sampleLoader.Indices.data());

	Renderer::Point2D p0 = Renderer::Point2D(INT2(0, 0), ColorRGBA(1.0f, 0.0f, 0.0f, 0.0f));
	Renderer::Point2D p1 = Renderer::Point2D(INT2(50, 0), ColorRGBA(1.0f, 0.0f, 0.0f, 0.0f));
	Renderer::Point2D p2 = Renderer::Point2D(INT2(25, 50), ColorRGBA(1.0f, 0.0f, 0.0f, 0.0f));

	mRenderer->SetTriangle(p0, p1, p2);


	// Math Test
	FLOAT4X4 View = Float4x4ViewMatrix(0, 0, 0);

	FLOAT4X4 Projection = Float4x4ProjectionMatrix(0.01f, 100.0f, DegreeToRadian(90.0f), 1.777f);

	//XMMATRIX xmProjection = XMMatrixPerspectiveFovLH(DegreeToRadian(90.0f), 1.777f, 0.01f, 100.0f);

	FLOAT4 tp0 = FLOAT4(0.0f, 5.0f, -1.0f, 1.0f);
	FLOAT4 tp1 = FLOAT4(2.5f, 0.0f, -1.0f, 1.0f);
	FLOAT4 tp2 = FLOAT4(-2.5f, 0.0f, -1.0f, 1.0f);

	// update from here 22/01/03 2:52 PM
	tp0 = Float4Multiply(tp0, View);
	tp1 = Float4Multiply(tp1, View);
	tp2 = Float4Multiply(tp2, View);

	tp0 = Float4Multiply(tp0, Projection);
	tp1 = Float4Multiply(tp1, Projection);
	tp2 = Float4Multiply(tp2, Projection);

	mRenderer->SetPixelNDC(tp0.x, tp0.y, ColorRGBA(1, 1, 1, 1));
	mRenderer->SetPixelNDC(tp1.x, tp1.y, ColorRGBA(1, 0, 0, 1));
	mRenderer->SetPixelNDC(tp2.x, tp2.y, ColorRGBA(0, 1, 0, 1));

}

void Engine::Update(float delta)
{
	mRenderer->ClearCanvas(ColorRGBA(0.0f, 0.0f, 0.0f, 0.0f));
	mRenderer->DrawScreen();


}

void Engine::Render(float delta)
{
	mRenderer->Present();

}

void Engine::Destroy()
{
}
