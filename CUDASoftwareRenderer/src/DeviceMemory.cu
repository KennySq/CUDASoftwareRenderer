#include<pch.h>
#include"DeviceMemory.cuh"

#include"Util.h"

__device__ void* gGlobalMemory = nullptr;

DeviceMemory::DeviceMemory(long long initSize)
	: mVirtual(nullptr), mOffset(0)
{
	
	hipDeviceSynchronize();

	hipDeviceProp_t deviceInfo{};
	int deviceAddr;
	
	hipError_t error = hipGetDevice(&deviceAddr);
	CUDAError(error);

	error = hipGetDeviceProperties(&deviceInfo, deviceAddr);
	CUDAError(error);
	hipDeviceSynchronize();

	size_t requestSize = deviceInfo.totalGlobalMem / 2;

	if (initSize != -1)
	{
		requestSize = initSize;
	}


	error = hipMalloc(reinterpret_cast<void**>(&gGlobalMemory), requestSize);
	CUDAError(error);

	if (error != NULL)
	{
		requestSize /= 2;
		error = hipMalloc(reinterpret_cast<void**>(&gGlobalMemory), requestSize);

		CUDAError(error);
	
		assert(error == NULL);
	}

	mVirtual = gGlobalMemory;
	mSize = requestSize;
	return;
}

DeviceMemory::~DeviceMemory()
{
	hipError_t error = hipFree(mVirtual);
	CUDAError(error);
}