#include "hip/hip_runtime.h"
#include<pch.h>
#include"Renderer.cuh"
#include"DIB.cuh"
#include"DeviceTexture.cuh"
#include"DeviceBuffer.cuh"
#include"ResourceManager.cuh"
#include"Geometry.cuh"
#include"3DMath.cuh"
#include"Util.h"

__device__ Renderer::Point2D* deviceDrawPoints = nullptr;
__device__ Renderer::Triangle2D* deviceTriangles = nullptr;

__global__ void KernelClearBitmap(void* target, unsigned int width, unsigned int height, ColorRGBA clearColor)
{
	unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	DWORD* asPixel = reinterpret_cast<DWORD*>(target);

	asPixel[index] = ConvertColorToDWORD(clearColor);
}

Renderer::Renderer(std::shared_ptr<DIB> dib, std::shared_ptr<ResourceManager> rs)
	: mCanvas(dib)
{
	unsigned int width = mCanvas->GetWidth();
	unsigned int height = mCanvas->GetHeight(); 

	mBuffer = rs->CreateTexture2D(width, height);

	mRenderPoints = new Point2D[width * height];
	mPointCount = 0;
	hipError_t error = hipMalloc(reinterpret_cast<void**>(&deviceDrawPoints), width * height * sizeof(Point2D));
	CUDAError(error);

	mRenderTriangles.resize(1024);

}

Renderer::~Renderer()
{
	Release();
}

void Renderer::SetPixel(int x, int y, const ColorRGBA& color)
{
	unsigned int width = mCanvas->GetWidth();
	unsigned int height = mCanvas->GetHeight();
	if (mPointCount > width * height)
	{
		return;
	}

	unsigned int _x = (width / 2 - x) - 1;
	unsigned int _y = (height / 2 - y) - 1;
	mRenderPoints[mPointCount] = Point2D(INT2(_x, _y), color);

	mPointCount++;
}

void Renderer::SetPixelNDC(float x, float y, const ColorRGBA& color)
{
	unsigned int width = mCanvas->GetWidth();
	unsigned int height = mCanvas->GetHeight();
	if (mPointCount > width * height)
	{
		return;
	}

	unsigned int _x = (width / 2 + (x * width)) - 1;
	unsigned int _y = (height / 2 + (y * height)) - 1;
	mRenderPoints[mPointCount] = Point2D(INT2(_x, _y), color);

	mPointCount++;
}

void Renderer::SetTriangle(const Point2D& p0, const Point2D& p1, const Point2D& p2)
{


	return;
}

void Renderer::OutText(int x, int y, std::string str)
{
	TextOutA(mCanvas->GetDC(), x, y, str.c_str(), str.size());
}

void Renderer::Start()
{

	unsigned int width = mCanvas->GetWidth();
	unsigned int height = mCanvas->GetHeight();

	hipError_t error = hipMalloc(reinterpret_cast<void**>(&deviceDrawPoints), width * height * sizeof(Point2D));
	CUDAError(error);
}

void Renderer::Update(float delta)
{

}

void Renderer::Render(float delta)
{
}

void Renderer::Release()
{
	hipFree(deviceDrawPoints);
	hipFree(deviceTriangles);
	delete[] mRenderPoints;
	mRenderPoints = nullptr;
}

void Renderer::ClearCanvas(const ColorRGBA& clearColor)
{
	unsigned int width = mCanvas->GetWidth();
	unsigned int height = mCanvas->GetHeight();

	void* texture = mBuffer->GetVirtual();

	dim3 block = dim3(32, 18, 1);
	dim3 grid = dim3(width / block.x, height / block.y, 1);

	//if (ColorRGBA(0, 0, 0, 0) == clearColor)
	//{
	//	return;
	//}

	KernelClearBitmap<<<grid, block>>>(texture, width, height, clearColor);

	hipDeviceSynchronize();
}

void Renderer::Present()
{
	mCanvas->CopyBuffer(mBuffer);
	mCanvas->Present();
}

inline __device__ void DeviceSetPixel(DWORD* buffer, unsigned int pointIndex, const ColorRGBA& color)
{
	buffer[pointIndex] = ConvertColorToDWORD(color);
}

inline __device__  void DeviceDrawLine(DWORD* buffer, const INT2& p0, const INT2& p1, unsigned int width, const ColorRGBA& color)
{

	INT2 from = p0;
	INT2 to = p1;

	auto sign = [](int dxy)
	{
		if (dxy < 0)
		{
			return -1;
		}
		else if (dxy > 0)
		{
			return 1;
		}
		else
		{
			return 0;
		}
	};

	int dx = to.x - from.x;
	int dy = to.y - from.y;

	int sx = sign(dx);
	int sy = sign(dy);

	dx = abs(dx);
	dy = abs(dy);

	int d = max(dx, dy);

	double r = d / 2;

	int x = from.x;
	int y = from.y;

	if (dx > dy)
	{
		for (int i = 0; i < d; i++)
		{
			unsigned int index = (y * width) + x;
			DeviceSetPixel(buffer, index, color);

			x += sx;
			r += dy;

			if (r >= dx)
			{
				y += sy;
				r -= dx;
			}
		}
	}
	else
	{
		for (int i = 0; i < d; i++)
		{
			unsigned int index = (y * width) + x;
			DeviceSetPixel(buffer, index, color);

			y += sy;
			r += dx;
			if (r >= dy)
			{
				x += sx;
				r -= dy;
			}

		}
	}


}

__global__ void KernelTransformVertices(DWORD* buffer, unsigned int width, unsigned int height, SampleVertex* vertices, VertexOutput* output, unsigned int* indices, unsigned int vertexCount, unsigned int indexCount, FLOAT4X4 Transform, FLOAT4X4 View, FLOAT4X4 Projection)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;//blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	unsigned int triThread = index * 3;

	auto clamp = [width, height](INT2& p)
	{
		if (p.x >= width)
		{
			p.x = width - 1;
		}
		else if (p.x < 0)
		{
			p.x = 0;
		}
		if (p.y >= height)
		{
			p.y = height - 1;
		}
		else if (p.y < 0)
		{
			p.y = 0;
		}
	};

	if (triThread + 2 >= vertexCount)
	{
		return;
	}

	unsigned int triIndex0 = indices[triThread];
	unsigned int triIndex1 = indices[triThread + 1];
	unsigned int triIndex2 = indices[triThread + 2];

	SampleVertex v0 = vertices[triIndex0];
	SampleVertex v1 = vertices[triIndex1];
	SampleVertex v2 = vertices[triIndex2];
	
	FLOAT4 position0 = FLOAT4(v0.Position.x, v0.Position.y, v0.Position.z, 1.0f);
	FLOAT4 position1 = FLOAT4(v1.Position.x, v1.Position.y, v1.Position.z, 1.0f);
	FLOAT4 position2 = FLOAT4(v2.Position.x, v2.Position.y, v2.Position.z, 1.0f);

	position0 = Float4Multiply(position0, Transform);
	position0 = Float4Multiply(position0, View);
	position0 = Float4Multiply(position0, Projection);
	
	position1 = Float4Multiply(position1, Transform);
	position1 = Float4Multiply(position1, View);
	position1 = Float4Multiply(position1, Projection);
	
	position2 = Float4Multiply(position2, Transform);
	position2 = Float4Multiply(position2, View);
	position2 = Float4Multiply(position2, Projection);

	//FLOAT4 normal0 = FLOAT4(v0.Normal.x, v0.Normal.y, v0.Normal.z, 1.0f);
	//FLOAT4 normal1 = FLOAT4(v1.Normal.x, v1.Normal.y, v1.Normal.z, 1.0f);
	//FLOAT4 normal2 = FLOAT4(v2.Normal.x, v2.Normal.y, v2.Normal.z, 1.0f);

	//FLOAT2 texcoord0 = FLOAT2(v0.Texcoord.x, v0.Texcoord.y);
	//FLOAT2 texcoord1 = FLOAT2(v1.Texcoord.x, v1.Texcoord.y);
	//FLOAT2 texcoord2 = FLOAT2(v2.Texcoord.x, v2.Texcoord.y);

	//output[triIndex0] = VertexOutput(position0, normal0, texcoord0);
	//output[triIndex1] = VertexOutput(position1, normal1, texcoord1);
	//output[triIndex2] = VertexOutput(position2, normal2, texcoord2);

	FLOAT3 ndcPosition0 = FLOAT3(position0.x / position0.w, position0.y / position0.w, position0.z / position0.w);
	FLOAT3 ndcPosition1 = FLOAT3(position1.x / position1.w, position1.y / position1.w, position1.z / position1.w);
	FLOAT3 ndcPosition2 = FLOAT3(position2.x / position2.w, position2.y / position2.w, position2.z / position2.w);

	INT2 point0 = NDCToScreen(ndcPosition0.x / ndcPosition0.z, ndcPosition0.y / ndcPosition0.z, width, height);
	INT2 point1 = NDCToScreen(ndcPosition1.x / ndcPosition1.z, ndcPosition1.y / ndcPosition1.z, width, height);
	INT2 point2 = NDCToScreen(ndcPosition2.x / ndcPosition2.z, ndcPosition2.y / ndcPosition2.z, width, height);

	clamp(point0);
	clamp(point1);
	clamp(point2);

	DeviceDrawLine(buffer, point0, point1, width, ColorRGBA(1, 0, 0, 0));
	DeviceDrawLine(buffer, point1, point2, width, ColorRGBA(0, 1, 0, 0));
	DeviceDrawLine(buffer, point2, point0, width, ColorRGBA(0, 0, 1, 0));
	
	// update from here 22/01/04 11:42 AM

	//const INT2 sampleLine1 = INT2(640, 360); // origin
	//const INT2 sampleLine2 = INT2(1279, 719);
	//const INT2 sampleLine3 = INT2(0, 0);
	//const INT2 sampleLine4 = INT2(-640+ (width/2), 360+ (height/2));
	//const INT2 sampleLine5 = INT2(1280, 0);
	//const INT2 sampleLine6 = INT2(640, 720);
	//const INT2 sampleLine7 = INT2(640, 0);

	//DeviceDrawLine(buffer, sampleLine1, sampleLine2, width, ColorRGBA(1, 0, 0, 0));
	//DeviceDrawLine(buffer, sampleLine1, sampleLine3, width, ColorRGBA(0, 1, 0, 0));
	//DeviceDrawLine(buffer, sampleLine1, sampleLine4, width, ColorRGBA(0, 0, 1, 0));
	//DeviceDrawLine(buffer, sampleLine1, sampleLine5, width, ColorRGBA(1, 1, 0, 0));
	//DeviceDrawLine(buffer, sampleLine1, sampleLine6, width, ColorRGBA(0, 1, 1, 0));
	//DeviceDrawLine(buffer, sampleLine1, sampleLine7, width, ColorRGBA(1, 0, 1, 0));
	//

	return;
}

__global__ void KernelDrawCallSetPixel(DWORD* buffer, Renderer::Point2D* drawPoints, unsigned int pixelCount, unsigned int width)
{
	unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	Renderer::Point2D pixel = drawPoints[index];
	unsigned int pointIndex = (pixel.Point.y * width) + pixel.Point.x;

	if (pointIndex >= pixelCount)
	{
		return;
	}

	buffer[pointIndex] = ConvertColorToDWORD(pixel.Color);

}



void Renderer::DrawScreen()
{
	unsigned int width = mCanvas->GetWidth();
	unsigned int height = mCanvas->GetHeight();

	dim3 block = dim3(32, 18, 1);
	dim3 grid = dim3(width / block.x, height / block.y, 1);
	
	void* buffer = mBuffer->GetVirtual();

	size_t copySize = width * height * sizeof(Point2D);
	hipError_t error = hipMemcpy(deviceDrawPoints, mRenderPoints, copySize, hipMemcpyHostToDevice);
	CUDAError(error);

	hipDeviceSynchronize();

	KernelDrawCallSetPixel << <grid, block >> > (CAST_PIXEL(buffer), deviceDrawPoints, width * height, width);

	hipDeviceSynchronize();
}

void Renderer::DrawTriangles(std::shared_ptr<DeviceBuffer> vertexBuffer, std::shared_ptr<DeviceBuffer> outputBuffer, std::shared_ptr<DeviceBuffer> indexBuffer, unsigned int vertexCount, unsigned int indexCount, const FLOAT4X4& transform, const FLOAT4X4& view, const FLOAT4X4& projection)
{
	unsigned int width = mCanvas->GetWidth();
	unsigned int height = mCanvas->GetHeight();

	void* buffer = mBuffer->GetVirtual();

	dim3 block = dim3(32, 1, 1);
	int left = indexCount % 3;
	dim3 grid = dim3((indexCount / block.x) + left, 1,1);
	
	if (grid.x == 0)
	{
		grid.x = 1;
	}

	SampleVertex* sampleVertices = reinterpret_cast<SampleVertex*>(vertexBuffer->GetVirtual());
	VertexOutput* outputVertices = reinterpret_cast<VertexOutput*>(outputBuffer->GetVirtual());

	unsigned int* indices = reinterpret_cast<unsigned int*>(indexBuffer->GetVirtual());

	KernelTransformVertices<<<grid,block>>>(CAST_PIXEL(buffer), width, height, sampleVertices, outputVertices, indices, vertexCount, indexCount, transform, view, projection);
	
	hipDeviceSynchronize();
}
