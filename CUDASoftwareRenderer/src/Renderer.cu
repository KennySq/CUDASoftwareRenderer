#include "hip/hip_runtime.h"
#include<pch.h>
#include"Renderer.cuh"
#include"DIB.cuh"
#include"DeviceTexture.cuh"
#include"DeviceBuffer.cuh"
#include"ResourceManager.cuh"
#include"Geometry.cuh"
#include"3DMath.cuh"
#include"Util.h"
#include"ShaderRegisterManager.cuh"

CUB_USE_COOPERATIVE_GROUPS;

__device__ Renderer::Point2D* deviceDrawPoints = nullptr;
__device__ ShaderRegisterManager* deviceRegisterManager = nullptr;

__global__ void KernelClearBitmap(void* target, unsigned int width, unsigned int height, ColorRGBA clearColor)
{
	unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	DWORD* asPixel = reinterpret_cast<DWORD*>(target);

	asPixel[index] = ConvertColorToDWORD(clearColor);
}

__global__ void KernelClearDepth(void* target, unsigned int width, unsigned int height, float v)
{
	unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	DWORD* asPixel = reinterpret_cast<DWORD*>(target);

	asPixel[index] = PackDepth(v);
}

Renderer::Renderer(std::shared_ptr<DIB> dib, std::shared_ptr<ResourceManager> rs)
	: mCanvas(dib)
{
	unsigned int width = mCanvas->GetWidth();
	unsigned int height = mCanvas->GetHeight();

	mBuffer = rs->CreateTexture2D(width, height);
	mDepth = rs->CreateTexture2D(width, height);

	mPointCount = 0;
	hipError_t error = hipMalloc(reinterpret_cast<void**>(&deviceDrawPoints), width * height * sizeof(Point2D));
	CUDAError(error);

	error = hipMalloc(reinterpret_cast<void**>(&deviceRegisterManager), sizeof(ShaderRegisterManager));
	CUDAError(error);

	mRasterizerBlock = dim3(32, 32, 1);
	mRasterizerGrid = dim3((width + mRasterizerBlock.x - 1) / mRasterizerBlock.x, (height + mRasterizerBlock.y - 1) / mRasterizerBlock.y, 1);

	hipDeviceProp_t deviceProp;
	error = hipGetDeviceProperties(&deviceProp, 0);
	CUDAError(error);

	std::cout << "Cooperative Launch Support: " << deviceProp.cooperativeLaunch << '\n';


}

Renderer::~Renderer()
{
}

__global__ void KernelDrawTexture(DWORD* texture, DWORD* buffer, int x, int y, unsigned int width)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x
		+ gridDim.x * gridDim.y * blockIdx.z;
	int threadId = blockId * (blockDim.x * blockDim.y)
		+ (threadIdx.y * blockDim.x) + threadIdx.x;

	unsigned int index = PointToIndex(INT2(x + threadId % width, y + threadId / width), 1280);

	buffer[index] = texture[threadId];
}

void Renderer::DrawTexture(std::shared_ptr<DeviceTexture> texture, int x, int y)
{
	void* ptr = texture->GetVirtual();

	dim3 block = dim3(32, 32, 1);
	
	dim3 grid = dim3(texture->GetWidth() / 32, texture->GetHeight() / 32, 1);
	KernelDrawTexture << <grid, block >> > (CAST_PIXEL(ptr), CAST_PIXEL(mBuffer->GetVirtual()), x, y, 64);

	return;
}

void Renderer::OutText(int x, int y, std::string str)
{
	RECT rect{};
	rect.right = x;
	rect.bottom = y;

	DrawTextA(mCanvas->GetHandleDC(), str.c_str(), str.size(), &rect, DT_BOTTOM | DT_INTERNAL | DT_NOCLIP);
}

__global__ void KernelGenerateDefaultTiles(Renderer::Tile* tiles, unsigned int tileCount)
{
	unsigned int tileIndex = (threadIdx.y * blockDim.x) + threadIdx.x;
	
	Renderer::Tile* tile = &tiles[tileIndex];
	tile = new (tile) Renderer::Tile;

	return;
}

void Renderer::Start()
{
	
	unsigned int width = mCanvas->GetWidth();
	unsigned int height = mCanvas->GetHeight();

	std::shared_ptr<ResourceManager> resourceManager = ResourceManager::GetInstance();
	
	unsigned int tileCount = mRasterizerGrid.x * mRasterizerGrid.y;
	mTileBuffer = resourceManager->CreateBuffer(sizeof(Tile), tileCount);
	dim3 tileGrid = dim3(1, 1, 1);
	dim3 tileBlock = dim3(mRasterizerGrid.x, mRasterizerGrid.y, 1);

	KernelGenerateDefaultTiles << <tileGrid, tileBlock >> > ((Renderer::Tile*)mTileBuffer->GetVirtual(), tileCount);
	hipDeviceSynchronize();
}

void Renderer::Update(float delta)
{

}

void Renderer::Render(float delta)
{
}

void Renderer::Release()
{
	hipFree(deviceDrawPoints);
	hipFree(deviceRegisterManager);
}

void Renderer::ClearCanvas(const ColorRGBA& clearColor)
{
	unsigned int width = mCanvas->GetWidth();
	unsigned int height = mCanvas->GetHeight();

	void* texture = mBuffer->GetVirtual();

	dim3 block = dim3(32, 18, 1);
	dim3 grid = dim3(width / block.x, height / block.y, 1);

	KernelClearBitmap << <grid, block >> > (texture, width, height, clearColor);

	hipDeviceSynchronize();
}

void Renderer::ClearDepth()
{
	unsigned int width = mCanvas->GetWidth();
	unsigned int height = mCanvas->GetHeight();

	void* depth = mDepth->GetVirtual();

	dim3 block = dim3(32, 18, 1);
	dim3 grid = dim3(width / block.x, height / block.y, 1);

	KernelClearDepth << <grid, block >> > (depth, width, height, 0.0f);
}

void Renderer::Present()
{
	mCanvas->CopyBuffer(mBuffer);
	mCanvas->Present();
}

inline __device__ void DeviceSetPixel(DWORD* buffer, unsigned int pointIndex, const ColorRGBA& color)
{
	buffer[pointIndex] = ConvertColorToDWORD(color);
}

inline __device__ void DeviceSetPixel(DWORD* buffer, unsigned int pointIndex, DWORD value)
{
	buffer[pointIndex] = value;
}

inline __device__ DWORD DeviceGetPixel(DWORD* buffer, unsigned int pointIndex)
{
	return buffer[pointIndex];
}

__device__ void DeviceGetTriangleDepth(const Renderer::Triangle& triangle, float& d0, float& d1, float& d2)
{
	const float fn0 = (100.0f + 0.01f) / (2.0f * (100.0f - 0.01f));
	const float fn1 = (-100.0f * 0.01f) / (100.0f - 0.01f);

	d0 = fn0 + (1.0f / HomogeneousToNDC(triangle.FragmentInput[0].Position).z) * fn1 + 0.5f;
	d1 = fn0 + (1.0f / HomogeneousToNDC(triangle.FragmentInput[1].Position).z) * fn1 + 0.5f;
	d2 = fn0 + (1.0f / HomogeneousToNDC(triangle.FragmentInput[2].Position).z) * fn1 + 0.5f;
}

__device__ float DeviceGetDepth(DWORD* depth, const INT2& point, unsigned int width)
{
	return depth[PointToIndex(point, width)];
}

__device__ void DeviceGetBarycentricAreas(const INT2& p4, const INT2& p0, const INT2& p1, const INT2& p2, float& u, float& v, float& w)
{
	INT2 ps0 = p1 - p0;
	INT2 ps1 = p2 - p0;
	INT2 ps2 = p4 - p0;

	FLOAT2 v0 = FLOAT2(ps0.x, ps0.y);
	FLOAT2 v1 = FLOAT2(ps1.x, ps1.y);
	FLOAT2 v2 = FLOAT2(ps2.x, ps2.y);

	float d00 = Float2Dot(v0, v0);
	float d01 = Float2Dot(v0, v1);
	float d11 = Float2Dot(v1, v1);
	float d20 = Float2Dot(v2, v0);
	float d21 = Float2Dot(v2, v1);

	float denom = (d00 * d11 - d01 * d01);

	v = (d11 * d20 - d01 * d21) / denom;
	w = (d00 * d21 - d01 * d20) / denom;
	u = 1.0f - v - w;
}

__device__ float DeviceInterpolateDepth(const Renderer::Triangle& triangle, float u, float v, float w)
{
	const float fn0 = (100.0f + 0.01f) / (2.0f * (100.0f - 0.01f));
	const float fn1 = (-100.0f * 0.01f) / (100.0f - 0.01f);

	FLOAT4 projected0 = triangle.FragmentInput[0].Position;
	FLOAT4 projected1 = triangle.FragmentInput[1].Position;
	FLOAT4 projected2 = triangle.FragmentInput[2].Position;

	FLOAT3 ndc0 = HomogeneousToNDC(projected0);
	FLOAT3 ndc1 = HomogeneousToNDC(projected1);
	FLOAT3 ndc2 = HomogeneousToNDC(projected2);

	float z = -(u * projected0.z + v * projected1.z + w * projected2.z);

	return fn0 + z * fn1 * 0.5f;
}

template<typename _Ty>
__device__ _Ty DeviceInterpolateByBarycentric(const _Ty& t0, const _Ty& t1, const _Ty& t2, float u, float v, float w)
{
	_Ty t = (t0 * u) + (t1 * v) + (t2 * w);

	return t;
}

__device__ VertexOutput DeviceInterpolateFragment(const VertexOutput& v0, const VertexOutput& v1, const VertexOutput& v2, float u, float v, float w)
{
	VertexOutput output;

	output.Position = DeviceInterpolateByBarycentric<FLOAT4>(v0.Position, v1.Position, v2.Position, u, v, w);
	output.Normal = DeviceInterpolateByBarycentric<FLOAT4>(v0.Normal, v1.Normal, v2.Normal, u, v, w);
	output.Texcoord = DeviceInterpolateByBarycentric<FLOAT2>(v0.Texcoord, v1.Texcoord, v2.Texcoord, u, v, w);

	return output;
}

__device__ FLOAT4 DeviceSampleTexture(void* texture, const FLOAT2& uv, unsigned int width, unsigned int height)
{
	DWORD* casted = CAST_PIXEL(texture);

	INT2 uvPoint = INT2(uv.x * width, uv.y * height);

	INT2 samplePoint = INT2(uvPoint.x, uvPoint.y);

	unsigned int index = PointToIndex(samplePoint, width);

	if (index >= width * height)
	{
		return;
	}

	ColorRGBA color = ConvertDWORDToColor(casted[index]);

	return FLOAT4(color.r, color.g, color.b, color.a);
}

__device__ FLOAT4 DeviceFragmentShader(ShaderRegisterManager* regManager, 
	const VertexOutput output[3], float u, float v, float w)
{
	VertexOutput interp = DeviceInterpolateFragment(output[0], output[1], output[2], u, v, w);
	ShaderRegisterManager::Register texture = regManager->Get(0, eRegisterType::REGISTER_TEXTURE);

	FLOAT2 uv = interp.Texcoord;

	FLOAT4 sampledTexture = DeviceSampleTexture(texture.Resource, uv, texture.Width, texture.Height);

	return sampledTexture;
}

__device__  void DeviceDrawLine(ShaderRegisterManager* regManager, DWORD* buffer, DWORD* depth,
	const INT2& p0, const INT2& p1,
	const Renderer::Triangle& triangle,
	unsigned int width, unsigned int height, const ColorRGBA& debugColor)
{
	INT2 from = p0;
	INT2 to = p1;

	Clamp<int>(from.x, 0, width);
	Clamp<int>(from.y, 0, height);
	Clamp<int>(to.x, 0, width);
	Clamp<int>(to.y, 0, height);

	auto sign = [](int dxy)
	{
		if (dxy < 0)
		{
			return -1;
		}
		else if (dxy > 0)
		{
			return 1;
		}
		else
		{
			return 0;
		}
	};

	int dx = to.x - from.x;
	int dy = to.y - from.y;

	int sx = sign(dx);
	int sy = sign(dy);

	dx = abs(dx);
	dy = abs(dy);

	int d = max(dx, dy);

	double r = (double)d / 2.0f;

	INT2 point = INT2(from.x, from.y);

	FLOAT3 ndc0 = HomogeneousToNDC(triangle.FragmentInput[0].Position);
	FLOAT3 ndc1 = HomogeneousToNDC(triangle.FragmentInput[1].Position);
	FLOAT3 ndc2 = HomogeneousToNDC(triangle.FragmentInput[2].Position);

	INT2 clip0 = NDCToClipSpace(ndc0, width, height);
	INT2 clip1 = NDCToClipSpace(ndc1, width, height);
	INT2 clip2 = NDCToClipSpace(ndc2, width, height);
	float u, v, w;

	if (dx > dy)
	{
		for (int i = 0; i <= d; i++)
		{
			unsigned int index = (point.y * width) + point.x;

			DeviceGetBarycentricAreas(point, clip0, clip1, clip2, u, v, w);

			float evalDepth = DeviceInterpolateDepth(triangle, u, v, w);

			int packed = PackDepth(evalDepth);

			atomicMax(reinterpret_cast<int*>(&depth[index]), packed);

			if (depth[index] == packed)
			{
				FLOAT4 result = DeviceFragmentShader(regManager, triangle.FragmentInput, u, v, w);

				DeviceSetPixel(buffer, index, ColorRGBA(result.x, result.y, result.z, result.w));
			}

			point.x += sx;
			r += dy;

			if (r >= dx)
			{
				point.y += sy;
				r -= dx;
			}
		}
	}
	else
	{
		for (int i = 0; i < d; i++)
		{
			unsigned int index = (point.y * width) + point.x;

			DeviceGetBarycentricAreas(point, clip0, clip1, clip2, u, v, w);
			float evalDepth = DeviceInterpolateDepth(triangle, u, v, w);


			int packed = PackDepth(evalDepth);

			atomicMax(reinterpret_cast<int*>(&depth[index]), packed);
			if (depth[index] == packed)
			{
				FLOAT4 result = DeviceFragmentShader(regManager, triangle.FragmentInput, u, v, w);

				DeviceSetPixel(buffer, index, ColorRGBA(result.x, result.y, result.z, result.w));
			}
			point.y += sy;
			r += dx;
			if (r >= dy)
			{
				point.x += sx;
				r -= dy;
			}

		}
	}
}

__device__ void DeviceFillBottomFlatTriangle(ShaderRegisterManager* regManager, DWORD* buffer, DWORD* depth,
	const INT2& p0, const INT2& p1, const INT2& p2, const Renderer::Triangle& triangle,
	unsigned int width, unsigned int height, unsigned int threadId,
	const ColorRGBA& debugColor)
{
	int p0yOffset = p0.y - threadId;

	float invSlope0 = ((p1.x - p0.x) / (float)(p1.y - p0.y));
	float invSlope1 = ((p2.x - p0.x) / (float)(p2.y - p0.y));

	float curx0 = p0.x - (invSlope0 * threadId);
	float curx1 = p0.x - (invSlope1 * threadId);

	int scanlineOffset = p0.y - p2.y;

	if (threadId > scanlineOffset || p0yOffset < p2.y || p0yOffset < 0)
	{
		return;
	}

	INT2 begin = INT2(curx0, p0yOffset);
	INT2 end = INT2(curx1, p0yOffset);

	DeviceDrawLine(regManager, buffer, depth, begin, end, triangle, width, height, debugColor);
}

__device__ void DeviceFillTopFlatTriangle(ShaderRegisterManager* regManager, DWORD* buffer, DWORD* depth,
	const INT2& p0, const INT2& p1, const INT2& p2, const Renderer::Triangle& triangle,
	unsigned int width, unsigned int height, unsigned int threadId,
	const ColorRGBA& debugColor)
{
	int p2yOffset = p2.y + threadId;

	float invSlope0 = ((p2.x - p0.x) / (float)(p2.y - p0.y));
	float invSlope1 = ((p2.x - p1.x) / (float)(p2.y - p1.y));

	float curx0 = p2.x + (invSlope0 * threadId);
	float curx1 = p2.x + (invSlope1 * threadId);

	int scanlineSize = p0.y - p2.y;

	if (threadId > scanlineSize || p2yOffset > p0.y || p2yOffset < 0)
	{
		return;
	}

	INT2 begin = INT2(curx0, p2yOffset);
	INT2 end = INT2(curx1, p2yOffset);
	DeviceDrawLine(regManager, buffer, depth, begin, end, triangle, width, height, debugColor);
}

__device__ void DeviceDrawFilledTriangle(ShaderRegisterManager* regManager, DWORD* buffer, DWORD* depth, const Renderer::Triangle& triangle,
	unsigned int width, unsigned int height, unsigned int threadId)
{
	FLOAT4 project0 = triangle.FragmentInput[0].Position;
	FLOAT4 project1 = triangle.FragmentInput[1].Position;
	FLOAT4 project2 = triangle.FragmentInput[2].Position;

	FLOAT3 ndc0 = HomogeneousToNDC(project0);
	FLOAT3 ndc1 = HomogeneousToNDC(project1);
	FLOAT3 ndc2 = HomogeneousToNDC(project2);

	INT2 cp0 = NDCToClipSpace(ndc0, width, height);
	INT2 cp1 = NDCToClipSpace(ndc1, width, height);
	INT2 cp2 = NDCToClipSpace(ndc2, width, height);

	auto sort = [&cp0, &cp1, &cp2]()
	{
		if (cp0.y < cp1.y)
		{
			Swap(cp0, cp1);
		}

		if (cp0.y < cp2.y)
		{
			Swap(cp0, cp2);
		}

		if (cp1.y < cp2.y)
		{
			Swap(cp1, cp2);
		}
	};

	sort();

	if (cp2.y == cp1.y)
	{
		DeviceFillBottomFlatTriangle(regManager, buffer, depth, cp0, cp1, cp2, triangle, width, height, threadId, ColorRGBA(1, 1, 1, 1));
	}

	else if (cp0.y == cp1.y)
	{
		DeviceFillTopFlatTriangle(regManager, buffer, depth, cp0, cp1, cp2, triangle, width, height, threadId, ColorRGBA(1, 1, 1, 1));
	}
	else
	{
		int midx = (cp0.x + ((float)(cp1.y - cp0.y) / (float)(cp2.y - cp0.y)) * (cp2.x - cp0.x));
		int midy = cp1.y;

		INT2 mid = INT2(midx, midy);

		DeviceFillTopFlatTriangle(regManager, buffer, depth, mid, cp1, cp2, triangle, width, height, threadId, ColorRGBA(1, 1, 1, 1));
		DeviceFillBottomFlatTriangle(regManager, buffer, depth, cp0, cp1, mid, triangle, width, height, threadId, ColorRGBA(1, 1, 1, 1));
	}
}

__device__ FLOAT3 DeviceGetSurfaceNormal(const FLOAT3& p0, const FLOAT3& p1, const FLOAT3& p2)
{
	FLOAT3 u = p1 - p0;
	FLOAT3 v = p2 - p0;

	return FLOAT3((u.y * v.z) - (u.z * v.y), (u.z * v.x) - (u.x * v.z), (u.x * v.y) - (u.y * v.x));
}

__global__ void KernelRasterize(ShaderRegisterManager* regManager, DWORD* buffer, DWORD* depth,
	unsigned int width, unsigned int height,
	Renderer::Triangle* triangles, unsigned int triangleCount, FLOAT3 viewPosition)
{
	unsigned int dispatchThreads = gridDim.y * gridDim.x * blockDim.x * blockDim.y;
	unsigned int threadPerTriangle = dispatchThreads / triangleCount;

	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x *blockDim.y)
		+ (threadIdx.y * blockDim.x) + threadIdx.x;

	unsigned int triIndex = (threadId / threadPerTriangle);

	if (triIndex >= triangleCount)
	{
		return;
	}

	Renderer::Triangle triangle = triangles[triIndex];

	FLOAT3 viewDir = triangle.SurfaceNormal - viewPosition;
	if (Float3Dot(Float3Normalize(viewDir), triangle.SurfaceNormal) < 0.0f)
	{
		return;
	}

	int scanlineIndex = threadId % threadPerTriangle;

	DeviceDrawFilledTriangle(regManager, buffer, depth, triangle, width, height, scanlineIndex);
}

__device__ AABB2D DeviceGetAABB(const INT2& c0, const INT2& c1, const INT2& c2)
{
	int maxX, maxY;
	int minX, minY;

	maxX = max(c0.x, c1.x);
	maxX = max(maxX, c2.x);

	minX = min(c0.x, c1.x);
	minX = min(minX, c2.x);

	maxY = max(c0.y, c1.y);
	maxY = max(maxY, c2.y);

	minY = min(c0.y, c1.y);
	minY = min(minY, c2.y);

	return AABB2D(INT2(minX, minY), INT2(maxX, maxY));
}

__device__ FLOAT3 DeviceGetBarycentric(const FLOAT4& p0, const FLOAT4& p1, const FLOAT4& p2)
{
	FLOAT3 ndc0 = HomogeneousToNDC(p0);
	FLOAT3 ndc1 = HomogeneousToNDC(p1);
	FLOAT3 ndc2 = HomogeneousToNDC(p2);

	FLOAT3 q = (ndc1 - ndc2) / 2;

	FLOAT3 p0toq = ndc0 - q;
	FLOAT3 p0top1 = ndc0 - ndc1;
	FLOAT3 p0top2 = ndc0 - ndc2;

	FLOAT3 result;

	float s0 = Float3Dot(p0toq, p0top1);
	float s1 = Float3Dot(p0top2, p0top2);
	float s2 = Float3Dot(p0top1, p0top1);
	float s3 = Float3Dot(p0top1, p0top2);
	float s4 = Float3Dot(p0toq, p0top2);

	result.x = ((s0 * s1) - (s4 * s3)) / ((s2 * s1) - (s3 * s3));
	result.y = ((s4 * s2) - (s0 * s3)) / ((s2 * s1) - (s3 * s3));

	return result;
}

__device__ void DeviceFillTriangle()
{

}

__global__ void KernelTileRasterize(unsigned int width, unsigned int height, Renderer::Tile* tiles, unsigned int tileCount, ShaderRegisterManager* regManager, DWORD* buffer, DWORD* depth, FLOAT3 viewPosition)
{
	unsigned int dispatchThreads = gridDim.y * gridDim.x * blockDim.x * blockDim.y;

	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadIndex = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	//int threadIndex = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	int threadX = blockIdx.x * blockDim.x + threadIdx.x;
	int threadY = blockIdx.y * blockDim.y + threadIdx.y;

	INT2 threadId = INT2(threadX, threadY);
	// 40 * 23
	if (blockId >= tileCount || threadIndex >= width * height)
	{
		return;
	}

	Renderer::Tile* tile = &tiles[blockId];
	size_t triangleCount = tile->Triangles.GetCount();
	Renderer::Triangle** triangles = (Renderer::Triangle**)tile->Triangles.GetData();

	for (int i = 0; i < triangleCount; i++)
	{
		Renderer::Triangle* tri = triangles[i];

		FLOAT4 p0 = tri->FragmentInput[0].Position;
		FLOAT4 p1 = tri->FragmentInput[1].Position;
		FLOAT4 p2 = tri->FragmentInput[2].Position;

		//FLOAT3 n0 = HomogeneousToNDC(p0);
		//FLOAT3 n1 = HomogeneousToNDC(p1);
		//FLOAT3 n2 = HomogeneousToNDC(p2);

		//INT2 c0 = NDCToClipSpace(n0, width, height);
		//INT2 c1 = NDCToClipSpace(n1, width, height);
		//INT2 c2 = NDCToClipSpace(n2, width, height);

		//float u, v, w;

		//DeviceGetBarycentricAreas(threadId, c0, c1, c2, u, v, w);

		//if (u == v == w)
		//{
		//	DeviceSetPixel(buffer, threadIndex, ConvertColorToDWORD(ColorRGBA(1.0f, 1.0f, 0.0f, 0.0f)));
		//}

		
	}

	__syncthreads();
	
	return;
}

__global__ void KernelTransformVertices(DWORD* buffer, DWORD* depth,
	Renderer::Triangle* triangles,
	unsigned int width, unsigned int height,
	SampleVertex* vertices, VertexOutput* output,
	unsigned int* indices, unsigned int vertexCount,
	unsigned int indexCount, FLOAT4X4 Transform, FLOAT4X4 MVP)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;//blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	unsigned int triThread = index * 3;

	if (triThread + 2 >= indexCount)
	{
		return;
	}

	unsigned int triIndex0 = indices[triThread];
	unsigned int triIndex1 = indices[triThread + 1];
	unsigned int triIndex2 = indices[triThread + 2];

	SampleVertex v0 = vertices[triIndex0];
	SampleVertex v1 = vertices[triIndex1];
	SampleVertex v2 = vertices[triIndex2];

	FLOAT4 position0 = FLOAT4(v0.Position.x, v0.Position.y, v0.Position.z, 1.0f);
	FLOAT4 position1 = FLOAT4(v1.Position.x, v1.Position.y, v1.Position.z, 1.0f);
	FLOAT4 position2 = FLOAT4(v2.Position.x, v2.Position.y, v2.Position.z, 1.0f);

	position0 = Float4Multiply(position0, MVP);
	position1 = Float4Multiply(position1, MVP);
	position2 = Float4Multiply(position2, MVP);

	FLOAT4 normal0 = FLOAT4(v0.Normal.x, v0.Normal.y, v0.Normal.z, 1.0f);
	FLOAT4 normal1 = FLOAT4(v1.Normal.x, v1.Normal.y, v1.Normal.z, 1.0f);
	FLOAT4 normal2 = FLOAT4(v2.Normal.x, v2.Normal.y, v2.Normal.z, 1.0f);

	FLOAT2 texcoord0 = FLOAT2(v0.Texcoord.x, v0.Texcoord.y);
	FLOAT2 texcoord1 = FLOAT2(v1.Texcoord.x, v1.Texcoord.y);
	FLOAT2 texcoord2 = FLOAT2(v2.Texcoord.x, v2.Texcoord.y);

	VertexOutput o0 = VertexOutput(position0, normal0, texcoord0);
	VertexOutput o1 = VertexOutput(position1, normal1, texcoord1);
	VertexOutput o2 = VertexOutput(position2, normal2, texcoord2);

	output[triThread] = o0;
	output[triThread + 1] = o1;
	output[triThread + 2] = o2;

	FLOAT3 ndcPosition0 = HomogeneousToNDC(position0);
	FLOAT3 ndcPosition1 = HomogeneousToNDC(position1);
	FLOAT3 ndcPosition2 = HomogeneousToNDC(position2);

	INT2 point0 = NDCToClipSpace(ndcPosition0, width, height);
	INT2 point1 = NDCToClipSpace(ndcPosition1, width, height);
	INT2 point2 = NDCToClipSpace(ndcPosition2, width, height);
	AABB2D aabb = DeviceGetAABB(point0, point1, point2);
	FLOAT3 barycentric = DeviceGetBarycentric(o0.Position, o1.Position, o2.Position);
	FLOAT3 surfaceNormal = DeviceGetSurfaceNormal(v0.Position, v1.Position, v2.Position);
	FLOAT4 wolrdSurfaceNormal = Float4Multiply(FLOAT4(surfaceNormal.x, surfaceNormal.y, surfaceNormal.z, 1.0f), Transform);

	surfaceNormal = FLOAT3(wolrdSurfaceNormal.x, wolrdSurfaceNormal.y, wolrdSurfaceNormal.z);

	triangles[index] = Renderer::Triangle(o0, o1, o2, aabb, barycentric, surfaceNormal);

	//DeviceDrawLine(buffer, depth, point0, point1, triangles[index], width, height, ColorRGBA(0, 0, 0, 0));
	//DeviceDrawLine(buffer, depth, point1, point2, triangles[index], width, height, ColorRGBA(0, 0, 0, 0));
	//DeviceDrawLine(buffer, depth, point2, point0, triangles[index], width, height, ColorRGBA(0, 0, 0, 0));

	return;
}

__global__ void KernelDrawCallSetPixel(DWORD* buffer, Renderer::Point2D* drawPoints, unsigned int pixelCount, unsigned int width)
{
	unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	Renderer::Point2D pixel = drawPoints[index];
	unsigned int pointIndex = (pixel.Point.y * width) + pixel.Point.x;

	if (pointIndex >= pixelCount)
	{
		return;
	}

	buffer[pointIndex] = ConvertColorToDWORD(pixel.Color);

}

__global__ void KernelGenerateTriangleBoundingTiles(dim3 tileGrid, dim3 rasterizerBlock, 
	unsigned int width, unsigned int height, Renderer::Tile* tiles, unsigned int tileCount, Renderer::Triangle* triangles, unsigned int triangleCount)
{
	unsigned int threadIndex = (blockDim.x * blockIdx.x) + threadIdx.x;

	if (threadIndex >= triangleCount)
	{
		return;
	}

	Renderer::Triangle* tri = &triangles[threadIndex];

	if (tri == nullptr)
	{
		return;
	}

	FLOAT4 p0 = tri->FragmentInput[0].Position;
	FLOAT4 p1 = tri->FragmentInput[1].Position;
	FLOAT4 p2 = tri->FragmentInput[2].Position;

	FLOAT3 n0 = HomogeneousToNDC(p0);
	FLOAT3 n1 = HomogeneousToNDC(p1);
	FLOAT3 n2 = HomogeneousToNDC(p2);

	INT2 c0 = NDCToClipSpace(n0, width, height);
	INT2 c1 = NDCToClipSpace(n1, width, height);
	INT2 c2 = NDCToClipSpace(n2, width, height);

	auto lambda_max = [](int f1, int f2) -> int
	{
		return f1 > f2 ? f1 : f2;
	};
	auto lambda_min = [](int f1, int f2) -> int
	{
		return f1 < f2 ? f1 : f2;
	};

	int maxY = lambda_max(lambda_max(c0.y, c1.y), c2.y);
	int maxX = lambda_max(lambda_max(c0.x, c1.x), c2.x);

	int minX = lambda_min(lambda_min(c0.x, c1.x), c2.x);
	int minY = lambda_min(lambda_min(c0.y, c1.y), c2.y);

	INT2 minPoint = INT2(minX, minY);
	INT2 maxPoint = INT2(maxX, maxY);

	INT2 minTileId = INT2(minPoint.x / rasterizerBlock.x, minPoint.y / rasterizerBlock.y);
	INT2 maxTileId = INT2(maxPoint.x / rasterizerBlock.x, maxPoint.y / rasterizerBlock.y);

	for (int i = minTileId.y; i < maxTileId.y; i++)
	{
		for (int j = minTileId.x; j < maxTileId.x; j++)
		{
			INT2 id = INT2(j, i);

			unsigned int tileIndex = (i * tileGrid.x) + j;
			
			if (tileIndex >= tileCount)
			{
				return;
			}

			Renderer::Tile* tile = &tiles[tileIndex];

			if (tile != nullptr)
			{
				tile->Triangles.Add((void*)tri, sizeof(Renderer::Triangle*));
			}
		}
	}

	return;
}

__global__ void KernelClearRasterizerBlocks(Renderer::Tile* tiles, unsigned int tileCount)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadIndex = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	int threadX = blockIdx.x * blockDim.x + threadIdx.x;
	int threadY = blockIdx.y * blockDim.y + threadIdx.y;

	if (threadIndex >= tileCount)
	{
		return;
	}

	tiles[threadIndex].Triangles.Clear();

	return;
}

void Renderer::DrawTriangles(std::shared_ptr<DeviceBuffer> vertexBuffer,
	std::shared_ptr<DeviceBuffer> indexBuffer,
	std::shared_ptr<DeviceBuffer> fragmentBuffer,
	std::shared_ptr<DeviceBuffer> triangleBuffer,
	unsigned int vertexCount, unsigned int indexCount,
	const FLOAT4X4& transform, const FLOAT4X4& view, const FLOAT4X4& projection)
{
	unsigned int width = mCanvas->GetWidth();
	unsigned int height = mCanvas->GetHeight();

	void* buffer = mBuffer->GetVirtual();
	void* depth = mDepth->GetVirtual();

	int totalThread = indexCount / 3;

	dim3 transformBlock = dim3(512, 1, 1);
	dim3 transformGrid = dim3(((totalThread + transformBlock.x - 1) / transformBlock.x), 1, 1);

	dim3 rasterBlock = dim3(24, 24, 1);
	dim3 rasterGrid = dim3((width + rasterBlock.x - 1) / rasterBlock.x, (height + rasterBlock.y - 1) / rasterBlock.y, 1);

	dim3 clearBlock = dim3(mRasterizerGrid.x, mRasterizerGrid.y, 1);
	dim3 clearGrid = dim3(1, 1, 1);

	if (transformGrid.x == 0)
	{
		transformGrid.x = 1;
	}

	SampleVertex* sampleVertices = reinterpret_cast<SampleVertex*>(vertexBuffer->GetVirtual());
	VertexOutput* outputVertices = reinterpret_cast<VertexOutput*>(fragmentBuffer->GetVirtual());
	Renderer::Triangle* triangles = reinterpret_cast<Renderer::Triangle*>(triangleBuffer->GetVirtual());

	unsigned int* indices = reinterpret_cast<unsigned int*>(indexBuffer->GetVirtual());

	float determView = Float4x4Determinant(view);
	FLOAT4X4 invView = Float4x4Multiply(view, determView);
	FLOAT3 viewPos = FLOAT3(invView._41, invView._42, invView._43);

	FLOAT4X4 mvp = Float4x4Multiply(transform, view);
	mvp = Float4x4Multiply(mvp, projection);

	KernelTransformVertices << <transformGrid, transformBlock >> >
		(CAST_PIXEL(buffer), CAST_PIXEL(depth),
			triangles, width, height,
			sampleVertices, outputVertices,
			indices, vertexCount, indexCount,
			transform, mvp);

	Tile* tileVirtual = reinterpret_cast<Tile*>(mTileBuffer->GetVirtual());
	unsigned int tileCount = mRasterizerGrid.x * mRasterizerGrid.y;
	KernelGenerateTriangleBoundingTiles << <transformGrid, transformBlock >> >
		(mRasterizerGrid, mRasterizerBlock, width, height, tileVirtual, tileCount, triangles, totalThread);
//	hipDeviceSynchronize();

	//void* args[] = { (void*)&width, (void*)&height, (void*)&tileVirtual, (void*)&tileCount, (void*)&deviceRegisterManager, (void*)CAST_PIXEL(buffer), (void*)CAST_PIXEL(depth), (void*)&viewPos };
	//hipLaunchCooperativeKernel(reinterpret_cast<const void*>(KernelTileRasterize), mRasterizerGrid, mRasterizerBlock, args, 0, nullptr);
	std::cout << hipGetErrorString(hipGetLastError()) << '\n';

	KernelTileRasterize <<<mRasterizerGrid, mRasterizerBlock>>>
		(width, height, tileVirtual, tileCount, 
			deviceRegisterManager, CAST_PIXEL(buffer), CAST_PIXEL(depth), viewPos);

	//hipDeviceSynchronize();
	std::cout << hipGetErrorString(hipGetLastError()) << '\n';

	KernelClearRasterizerBlocks << <clearGrid, clearBlock >> > (tileVirtual, tileCount);
	//hipDeviceSynchronize();

	//KernelRasterize << <rasterGrid, rasterBlock >> >
	//	(deviceRegisterManager, CAST_PIXEL(buffer), CAST_PIXEL(depth),
	//		width, height, triangles, totalThread, viewPos);
}

__global__ void KernelSetRegister(ShaderRegisterManager* regManager, void* ptr,
	unsigned int index, unsigned int width, unsigned int height, eRegisterType regType)
{
	if (threadIdx.x == 0)
	{
		regManager->Set(ptr, index, width, height, regType);
	}

}

void Renderer::BindTexture(std::shared_ptr<DeviceTexture> texture, unsigned int index)
{
	assert(texture != nullptr);

	void* ptr = texture->GetVirtual();
	unsigned int width = texture->GetWidth();
	unsigned int height = texture->GetHeight();

	KernelSetRegister << <1, 1 >> > (deviceRegisterManager, ptr, index, width, height, eRegisterType::REGISTER_TEXTURE);

	return;
}
