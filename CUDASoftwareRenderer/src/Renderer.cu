#include "hip/hip_runtime.h"
#include<pch.h>
#include"Renderer.cuh"
#include"DIB.cuh"
#include"DeviceTexture.cuh"
#include"Color.cuh"
#include"ResourceManager.cuh"

__global__ void KernelClearBitmap(void* target, unsigned int width, unsigned int height, ColorRGBA clearColor)
{
	unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	DWORD* asPixel = reinterpret_cast<DWORD*>(target);

	asPixel[index] = ConvertColorToDWORD(clearColor);
}

Renderer::Renderer(std::shared_ptr<DIB> dib, std::unique_ptr<ResourceManager>&& rs)
	: mCanvas(dib)
{
	mBuffer = rs->CreateTexture2D(dib->GetWidth(), dib->GetHeight());
}

void Renderer::ClearCanvas(ColorRGBA clearColor)
{
	unsigned int width = mCanvas->GetWidth();
	unsigned int height = mCanvas->GetHeight();

	void* texture = mBuffer->GetVirtual();

	dim3 block = dim3(32, 18, 1);
	dim3 grid = dim3(width / block.x, height / block.y, 1);

	KernelClearBitmap<<<grid, block>>>(texture, width, height, clearColor);
}

void Renderer::Present()
{
	mCanvas->CopyBuffer(mBuffer);
	mCanvas->Present();
}


